#include "select_conv_cuda.cuh"


void select_im2col_cuda(Tensor data_im,
                            const int channels, const int height,
                            const int width, const int ksize_h,
                            const int ksize_w, const int pad_h, const int pad_w,
                            const int stride_h, const int stride_w,
                            const int dilation_h, const int dilation_w,
                            const int parallel_imgs,
                            Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels = channels * height_col * width_col * parallel_imgs;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "select_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        // const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        select_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, height, width, ksize_h,
            ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
            parallel_imgs, channels, height_col, width_col, data_col_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}


void select_col2im_cuda(
    Tensor data_col, const int channels, const int height,
    const int width, const int ksize_h, const int ksize_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int dilation_h, const int dilation_w,
    const int parallel_imgs, Tensor grad_im) {
  // todo: make sure parallel_imgs is passed in correctly
  int height_col =
      (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / stride_h + 1;
  int width_col =
      (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / stride_w + 1;
  int num_kernels =
      channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
//   int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "deformable_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        // const scalar_t *data_offset_ = data_offset.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        select_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, channels, height, width,
            ksize_h, ksize_w, pad_h, pad_w, stride_h, stride_w, dilation_h,
            dilation_w, parallel_imgs,
            height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

